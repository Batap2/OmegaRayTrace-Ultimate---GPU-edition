#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION

#include <iostream>
#include <string>

#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include "deps/glm/ext.hpp"

#include "deps/imgui/imgui.h"
#include "deps/imgui/imgui_impl_glfw.h"
#include "deps/imgui/imgui_impl_opengl3.h"


#include "Mesh.h"
#include "camera.h"
#include "GUI.h"
#include "ShaderUtils.h"
#include "SceneOperations.h"

#include "shaders.h"
#include "transform.h"
#include "globals.h"

// CUDA - Merging
#include "GPURenderer.h"

// Gestion des erreurs CUDA
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
    if (button == GLFW_MOUSE_BUTTON_RIGHT && action == GLFW_PRESS){
        rclickHolded = true;
        glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
    }
    if (button == GLFW_MOUSE_BUTTON_RIGHT && action == GLFW_RELEASE){
        rclickHolded = false;
        glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
    }
}

void keyboard(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action == GLFW_PRESS){
        switch(key){
            case GLFW_KEY_ESCAPE: 
                exit(0) ;
                break ;
            case GLFW_KEY_R:
                mainCamera.cameraPos = eyeinit ;
                mainCamera.cameraUp = upinit ;
                amount = amountinit ;
                transop = view2 ;
                sx = sy = 1.0 ; 
                tx = ty = 0.0 ; 
                break ;
            case GLFW_KEY_A:
                leftHolded = true;
                break;
            case GLFW_KEY_D:
                rightHolded = true;
                break;
            case GLFW_KEY_W:
                upHolded = true;
                break;
            case GLFW_KEY_S:
                 downHolded = true;
                break;
            case GLFW_KEY_Q:
                aHolded = true;
                break;
            case GLFW_KEY_E:
                eHolded = true;
                break;
            case GLFW_KEY_TAB:
                showMenus = !showMenus;
                break;
        }
    } else if (action == GLFW_RELEASE)
    {
        switch(key){
            case GLFW_KEY_A:
                leftHolded = false;
                break;
            case GLFW_KEY_D:
                rightHolded = false;
                break;
            case GLFW_KEY_W:
                upHolded = false;
                break;
            case GLFW_KEY_S:
                downHolded = false;
                break;
            case GLFW_KEY_Q:
                aHolded = false;
                break;
            case GLFW_KEY_E:
                eHolded = false;
                break;
        }
    }
}

void mouse_callback(GLFWwindow* window, double xpos, double ypos)
{
    if(rclickHolded)
    {
        float xoffset = xpos - lastX;
        float yoffset = lastY - ypos;
        lastX = xpos;
        lastY = ypos;

        float sensitivity = 0.1f;
        xoffset *= sensitivity;
        yoffset *= sensitivity;

        yaw   += xoffset;
        pitch += yoffset;

        if(pitch > 89.0f)
            pitch = 89.0f;
        if(pitch < -89.0f)
            pitch = -89.0f;

        glm::vec3 direction;
        direction.x = cos(glm::radians(yaw)) * cos(glm::radians(pitch));
        direction.y = sin(glm::radians(pitch));
        direction.z = sin(glm::radians(yaw)) * cos(glm::radians(pitch));

        mainCamera.cameraDirection = glm::normalize(direction);
        mainCamera.cameraRight = glm::normalize(glm::cross(mainCamera.cameraDirection, upinit));
        mainCamera.cameraUp = glm::cross(mainCamera.cameraRight, mainCamera.cameraDirection);
    } else {
        lastX = xpos;
        lastY = ypos;
    }
}

static void scroll_callback(GLFWwindow* window, double xoffset, double yoffset){
    cameraSpeed += yoffset * 0.005;
    cameraSpeed = fmax(0.0f, cameraSpeed);
}

void manageInput()
{
    if(rclickHolded)
    {
        if(leftHolded){
            mainCamera.cameraPos -= cameraSpeed * mainCamera.cameraRight;
        }
        if(rightHolded){
            mainCamera.cameraPos += cameraSpeed * mainCamera.cameraRight;
        }
        if(upHolded){
            mainCamera.cameraPos += cameraSpeed * mainCamera.cameraDirection;
        }
        if(downHolded){
            mainCamera.cameraPos -= cameraSpeed * mainCamera.cameraDirection;
        }
        if(aHolded){
            mainCamera.cameraPos -= cameraSpeed * upinit;
        }
        if(eHolded){
            mainCamera.cameraPos += cameraSpeed * upinit;
        }
    }
}

void display() {
    glClearColor(0, 0.2, 0, 0);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    modelview = glm::lookAt(mainCamera.cameraPos,mainCamera.cameraPos + mainCamera.cameraDirection, mainCamera.cameraUp);

    glUniformMatrix4fv(projectionLoc, 1, GL_FALSE, &projection[0][0]);
    glUniformMatrix4fv(modelviewLoc, 1, GL_FALSE, &modelview[0][0]);

    glUniform3fv(camPosLoc, 1, glm::value_ptr(mainCamera.cameraPos));

    const GLfloat white[4] = {1, 1, 1, 1};
    const GLfloat black[4] = {0, 0, 0, 0};

    glUniformMatrix4fv(modelviewLoc, 1, GL_FALSE, &modelview[0][0]);

    for(Mesh* meshP : scene_meshes)
    {
        glBindVertexArray(meshP->VAO);
        glBindTexture(GL_TEXTURE_2D, meshP->diffuse_texture_id);

        glDrawElements(GL_TRIANGLES, meshP->indicies.size(), GL_UNSIGNED_INT, 0);
    }

}

int main(int argc, char* argv[]){


    // int nx = 1200;
    // int ny = 600;
    // int tx = 8;
    // int ty = 8;

    // std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    // std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    // int num_pixels = nx*ny;
    // size_t fb_size = num_pixels*sizeof(vec3);

    // // allocate FB
    // vec3 *fb;
    // checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // clock_t start, stop;
    // start = clock();
    // // Render our buffer
    // dim3 blocks(nx/tx+1,ny/ty+1);
    // dim3 threads(tx,ty);
    // GPURenderer::render<<<blocks, threads>>>(fb, nx, ny);
    // checkCudaErrors(hipGetLastError());
    // checkCudaErrors(hipDeviceSynchronize());
    // stop = clock();
    // double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    // std::cerr << "took " << timer_seconds << " seconds.\n";

    // // Output FB as Image
    // std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    // for (int j = ny-1; j >= 0; j--) {
    //     for (int i = 0; i < nx; i++) {
    //         size_t pixel_index = j*nx + i;
    //         int ir = int(255.99*fb[pixel_index][0]);
    //         int ig = int(255.99*fb[pixel_index][1]);
    //         int ib = int(255.99*fb[pixel_index][2]);
    //         std::cout << ir << " " << ig << " " << ib << "\n";
    //     }
    // }

    // checkCudaErrors(hipFree(fb));

    // Initialise GLFW and GLEW; and parse path from command line  
    GLFWwindow* window;

    if (!glfwInit()) return -1;
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3); 
    window = glfwCreateWindow(window_width, window_height, "Scene Viewer", NULL, NULL);
    glfwMakeContextCurrent(window);
    glfwSwapInterval(1);

    if (glewInit() != GLEW_OK) std::cout << "ERROR : glewInit\n ";
    std::cout << glGetString(GL_VERSION) << "\n"; 

    // Initialize ImGui 
    const char* glsl_version = "#version 330";
    ImGui::CreateContext();
    ImGui_ImplGlfw_InitForOpenGL(window, true); 
    ImGui_ImplOpenGL3_Init(glsl_version);
    ImGui::StyleColorsDark();

    // Initialise all variable initial values
    ShaderUtils::init_shaders();

    glEnable(GL_DEPTH_TEST);
    glfwSetWindowSizeCallback(window, ShaderUtils::reshape);

    glfwSetKeyCallback(window, keyboard);
    glfwSetMouseButtonCallback(window, mouse_button_callback);
    glfwSetInputMode(window, GLFW_STICKY_KEYS, GLFW_TRUE);

    glfwSetCursorPosCallback(window, mouse_callback);
    glfwSetScrollCallback(window, scroll_callback);


    SceneOperations::initSceneLights();
    SceneOperations::openFile("data/cube3.fbx");

    ShaderUtils::reshape(window, window_width, window_height);
    ShaderUtils::sendLightsToShaders();

    std::vector<Mesh*> scene_meshes_check = scene_meshes;

    while (!glfwWindowShouldClose(window))
    {
        manageInput();

        display();

        if(showMenus){
            GUI::draw(window);
        }

        glfwSwapBuffers(window);
        glfwPollEvents();

    }

    SceneOperations::destroyScene();
    mesh.destroy_buffers();
    glfwTerminate();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();
    return 0;
}
